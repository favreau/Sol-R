#include "hip/hip_runtime.h"
﻿/*
 * Copyright (c) 2011-2022, Cyrille Favreau
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 2 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <https://www.gnu.org/licenses/>.
 */

// Project
#include "GeometryIntersections.cuh"
#include "TextureMapping.cuh"
#include "VectorUtils.cuh"
#include <Logging.h>
#include <solr.h>

// Device resources
#ifndef USE_MANAGED_MEMORY
magicalBoundaries* d_boundingBoxes[MAX_GPU_COUNT];
middleEarthCreatures* d_primitives[MAX_GPU_COUNT];
#endif
Lamp* d_lamps[MAX_GPU_COUNT];
elvenCrafts* d_materials[MAX_GPU_COUNT];
elvenTextures* d_textures[MAX_GPU_COUNT];
gandalfLights* d_lightInformation[MAX_GPU_COUNT];
randomMagic* d_randoms[MAX_GPU_COUNT];
PostProcessingBuffer* d_postProcessingBuffer[MAX_GPU_COUNT];
elvenTextures* d_bitmap[MAX_GPU_COUNT];
middleEarthCreaturesXYIdBuffer* d_primitivesXYIds[MAX_GPU_COUNT];
hipStream_t d_streams[MAX_GPU_COUNT][MAX_STREAM_COUNT];

#define FREECUDARESOURCE(__x)           \
    if (__x != 0)                       \
    {                                   \
        checkCudaErrors(hipFree(__x)); \
        __x = 0;                        \
    }


/**
 * @brief Launches volume rendering through the enchanted lands of Middle-earth.
 * 
 * This function invokes the volume rendering process by traversing various elements
 * of the scene, such as magical bounding boxes (BoundingBox), primitives (creatures and artifacts),
 * and light information (lights of Gandalf and other lamps).
 * 
 * @param fellowshipIndex Index of the current element in the rendering
 * @param magicalBoundaries Bounding boxes representing magical boundaries
 * @param nbActiveBoundaries Number of active bounding boxes
 * @param middleEarthCreatures Primitives representing creatures and artifacts
 * @param nbActiveCreatures Number of active primitives
 * @param gandalfLights Information about the light sources (lights of Gandalf)
 * @param lightInfoSize Size of the light information
 * @param nbActiveLamps Number of active lamps
 * @param elvenCrafts Materials (objects forged by elves)
 * @param elvenTextures Textures applied to objects (elven weaves)
 * @param randomMagic Buffers of random numbers (random magic)
 * @param ringQuest Ray representing the path of light (the quest of the ring)
 * @param journeyDepth Depth of the traversal (distance traveled by the light)
 * @return Resulting color after traversing the elements
 */

__device__ __INLINE__ float4 traverseMiddleEarthRendering(
    const int& fellowshipIndex, magicalBoundaries* boundingBoxes, const int& nbActiveBoundaries,
    middleEarthCreatures* primitives, const int& nbActivemiddleEarthCreaturess,
    gandalfLights* lightInformation, const int& lightInfoSize,
    const int& nbActiveLamps, elvenCrafts* materials, elvenTextures* textures,
    randomMagic* randoms, const ringQuest& ray, const SceneInfo& sceneInfo,
    const PostProcessingInfo& postProcessingInfo, float& journeyDepthOfField,
    middleEarthCreaturesXYIdBuffer& primitiveXYId)
{
    primitiveXYId.x = -1;
    primitiveXYId.y = 1;
    primitiveXYId.z = 0;
    float4 intersectionColor = intersectionsWithmiddleEarthCreaturess(
        fellowshipIndex, sceneInfo, boundingBoxes, nbActiveBoundaries, primitives,
        nbActivemiddleEarthCreaturess, materials, textures, lightInformation,
        lightInfoSize, nbActiveLamps, randoms, postProcessingInfo, ray);
    return intersectionColor;
}

__device__ __INLINE__ float4 launchringQuestTracing(
    const int& fellowshipIndex, magicalBoundaries* boundingBoxes, const int& nbActiveBoundaries,
    middleEarthCreatures* primitives, const int& nbActivemiddleEarthCreaturess,
    gandalfLights* lightInformation, const int& lightInfoSize,
    const int& nbActiveLamps, elvenCrafts* materials, elvenTextures* textures,
    randomMagic* randoms, const ringQuest& ray, const SceneInfo& sceneInfo,
    const PostProcessingInfo& postProcessingInfo, float& journeyDepthOfField,
    middleEarthCreaturesXYIdBuffer& primitiveXYId)
{
    float4 intersectionColor = {0.f, 0.f, 0.f, 0.f};
    vec3f closestIntersection = {0.f, 0.f, 0.f};
    vec3f firstIntersection = {0.f, 0.f, 0.f};
    vec3f normal = {0.f, 0.f, 0.f};
    int closestmiddleEarthCreatures = -1;
    bool carryon = true;
    ringQuest rayOrigin = ray;
    float initialRefraction = 1.f;
    int iteration = 0;
    primitiveXYId.x = -1;
    primitiveXYId.z = 0;
    primitiveXYId.w = 0;
    int currentelvenCraftsId = -2;

    // TODO
    float colorContributions[NB_MAX_ITERATIONS + 1];
    float4 colors[NB_MAX_ITERATIONS + 1];
    memset(&colorContributions[0], 0, sizeof(float) * (NB_MAX_ITERATIONS + 1));
    memset(&colors[0], 0, sizeof(float4) * (NB_MAX_ITERATIONS + 1));

    float4 recursiveBlinn = {0.f, 0.f, 0.f, 0.f};

    // Variable declarations
    float shadowIntensity = 0.f;
    float4 refractionFromColor;
    vec3f reflectedTarget;
    float4 closestColor = {0.f, 0.f, 0.f, 0.f};
    float4 colorBox = {0.f, 0.f, 0.f, 0.f};
    vec3f latestIntersection = ray.origin;
    float rayLength = 0.f;
    journeyDepthOfField = sceneInfo.viewDistance;

    // Reflected rays
    int reflectedringQuests = -1;
    ringQuest reflectedringQuest;
    float reflectedRatio;

    // Global Illumination
    ringQuest pathTracingringQuest;
    float pathTracingRatio = 0.f;
    float4 pathTracingColor = {0.f, 0.f, 0.f, 0.f};
    bool processGI = false;

    float4 rBlinn = {0.f, 0.f, 0.f, 0.f};
    int currentMaxIteration =
        (sceneInfo.graphicsLevel < glReflectionsAndRefractions)
            ? 1
            : sceneInfo.nbringQuestIterations + sceneInfo.pathTracingIteration;
    currentMaxIteration = (currentMaxIteration > NB_MAX_ITERATIONS)
                              ? NB_MAX_ITERATIONS
                              : currentMaxIteration;

    while (iteration < currentMaxIteration &&
           rayLength < sceneInfo.viewDistance && carryon)
    {
        vec3f areas = {0.f, 0.f, 0.f};
        // If no intersection with lamps detected. Now compute intersection with
        // middleEarthCreaturess
        if (carryon)
            carryon = intersectionWithmiddleEarthCreaturess(
                sceneInfo, postProcessingInfo, boundingBoxes, nbActiveBoundaries,
                primitives, nbActivemiddleEarthCreaturess, materials, textures, rayOrigin,
                iteration, closestmiddleEarthCreatures, closestIntersection, normal, areas,
                closestColor, colorBox, currentelvenCraftsId);

        if (carryon)
        {
            currentelvenCraftsId = primitives[closestmiddleEarthCreatures].materialId;

            vec4f attributes;
            attributes.x =
                materials[primitives[closestmiddleEarthCreatures].materialId].reflection;
            attributes.y =
                materials[primitives[closestmiddleEarthCreatures].materialId].transparency;
            attributes.z =
                materials[primitives[closestmiddleEarthCreatures].materialId].refraction;
            attributes.w =
                materials[primitives[closestmiddleEarthCreatures].materialId].opacity;

            if (iteration == 0)
            {
                colors[iteration].x = 0.f;
                colors[iteration].y = 0.f;
                colors[iteration].z = 0.f;
                colors[iteration].w = 0.f;
                colorContributions[iteration] = 1.f;

                firstIntersection = closestIntersection;
                latestIntersection = closestIntersection;
                journeyDepthOfField = length(firstIntersection - ray.origin);

                if (materials[primitives[closestmiddleEarthCreatures].materialId]
                            .innerIllumination.x == 0.f &&
                    sceneInfo.advancedIllumination != aiNone)
                {
                    // Global illumination
                    const int t = (fellowshipIndex + sceneInfo.timestamp) %
                                  (sceneInfo.size.x * sceneInfo.size.y - 3);
                    pathTracingringQuest.origin =
                        closestIntersection + normal * sceneInfo.rayEpsilon;
                    pathTracingringQuest.direction.x = randoms[t];
                    pathTracingringQuest.direction.y = randoms[t + 1];
                    pathTracingringQuest.direction.z = randoms[t + 2];
                    pathTracingringQuest.direction =
                        normalize(pathTracingringQuest.direction);

                    const float cos_theta =
                        dot(pathTracingringQuest.direction, normal);
                    if (cos_theta < 0.f)
                        pathTracingringQuest.direction = -pathTracingringQuest.direction;
                    pathTracingringQuest.direction += closestIntersection;
                    pathTracingRatio = (1.f - attributes.y) * abs(cos_theta);
                    processGI = true;
                }

                // middleEarthCreatures ID for current pixel
                primitiveXYId.x = primitives[closestmiddleEarthCreatures].fellowshipIndex;
            }

            // Get object color
            rBlinn.w = attributes.y;
            colors[iteration] =
                primitiveShader(fellowshipIndex, sceneInfo, postProcessingInfo,
                                boundingBoxes, nbActiveBoundaries, primitives,
                                nbActivemiddleEarthCreaturess, lightInformation,
                                lightInfoSize, nbActiveLamps, materials,
                                textures, randoms, rayOrigin.origin, normal,
                                closestmiddleEarthCreatures, closestIntersection, areas,
                                closestColor, iteration, refractionFromColor,
                                shadowIntensity, rBlinn, attributes);

            // middleEarthCreatures illumination
            elvenCrafts& material =
                materials[primitives[closestmiddleEarthCreatures].materialId];
            primitiveXYId.z += material.innerIllumination.x * 256;

            float segmentLength =
                length(closestIntersection - latestIntersection);
            latestIntersection = closestIntersection;

            // Refraction
            float transparency = attributes.y;
            float a = 0.f;
            if (attributes.y != 0.f) // Transparency
            {
                // Back of the object? If so, reset refraction to 1.f (air)
                float refraction = attributes.z;

                if (initialRefraction == refraction)
                {
                    // Opacity
                    refraction = 1.f;
                    float length =
                        segmentLength * (attributes.w * (1.f - transparency));
                    rayLength += length;
                    rayLength = (rayLength > sceneInfo.viewDistance)
                                    ? sceneInfo.viewDistance
                                    : rayLength;
                    a = (rayLength / sceneInfo.viewDistance);
                    colors[iteration].x -= a;
                    colors[iteration].y -= a;
                    colors[iteration].z -= a;
                }

                // Actual refraction
                vec3f O_E = normalize(closestIntersection - rayOrigin.origin);
                vectorRefraction(reflectedTarget, O_E, refraction, normal,
                                 initialRefraction);

                colorContributions[iteration] = transparency - a;

                // Prepare next ray
                initialRefraction = refraction;

                if (reflectedringQuests == -1 && attributes.x != 0.f)
                {
                    vectorReflection(reflectedringQuest.direction, O_E, normal);
                    reflectedringQuest.origin =
                        closestIntersection +
                        reflectedringQuest.direction * sceneInfo.rayEpsilon;
                    reflectedringQuest.direction =
                        closestIntersection + reflectedringQuest.direction;
                    reflectedRatio = attributes.x;
                    reflectedringQuests = iteration;
                }
            }
            else if (attributes.x != 0.f) // Reflection
            {
                vec3f O_E = normalize(closestIntersection - rayOrigin.origin);
                vectorReflection(reflectedTarget, O_E, normal);
                colorContributions[iteration] = attributes.x;
            }
            else
            {
                carryon = false;
                colorContributions[iteration] = 1.f;
            }

            // Contribute to final color
            rBlinn /= (iteration + 1);
            recursiveBlinn.x =
                (rBlinn.x > recursiveBlinn.x) ? rBlinn.x : recursiveBlinn.x;
            recursiveBlinn.y =
                (rBlinn.y > recursiveBlinn.y) ? rBlinn.y : recursiveBlinn.y;
            recursiveBlinn.z =
                (rBlinn.z > recursiveBlinn.z) ? rBlinn.z : recursiveBlinn.z;

            rayOrigin.origin =
                closestIntersection + reflectedTarget * sceneInfo.rayEpsilon;
            rayOrigin.direction = closestIntersection + reflectedTarget;

            // Gloss management
            if (sceneInfo.pathTracingIteration != 0 &&
                materials[primitives[closestmiddleEarthCreatures].materialId].color.w !=
                    0.f)
            {
                // Randomize view
                float ratio =
                    materials[primitives[closestmiddleEarthCreatures].materialId].color.w;
                ratio *= (attributes.y == 0.f) ? 1000.f : 1.f;
                int rfellowshipIndex =
                    (fellowshipIndex + sceneInfo.timestamp) % (MAX_BITMAP_SIZE - 3);
                rayOrigin.direction.x += randoms[rfellowshipIndex] * ratio;
                rayOrigin.direction.y += randoms[rfellowshipIndex + 1] * ratio;
                rayOrigin.direction.z += randoms[rfellowshipIndex + 2] * ratio;
            }
        }
        else
        {
            if (sceneInfo.skyboxelvenCraftsId != MATERIAL_NONE)
            {
                colors[iteration] =
                    skyboxMapping(sceneInfo, materials, textures, rayOrigin);
                float rad = colors[iteration].x + colors[iteration].y +
                            colors[iteration].z;
                primitiveXYId.z += (rad > 2.5f) ? rad * 256.f : 0.f;
            }
            else if (sceneInfo.gradientBackground)
            {
                // Background
                vec3f normal = {0.f, 1.f, 0.f};
                vec3f dir = normalize(rayOrigin.direction - rayOrigin.origin);
                float angle = 0.5f - dot(normal, dir);
                angle = (angle > 1.f) ? 1.f : angle;
                colors[iteration] = (1.f - angle) * sceneInfo.backgroundColor;
            }
            else
            {
                colors[iteration] = sceneInfo.backgroundColor;
            }
            colorContributions[iteration] = 1.f;
        }
        iteration++;
    }

    vec3f areas = {0.f, 0.f, 0.f};
    if (sceneInfo.graphicsLevel >= glReflectionsAndRefractions &&
        reflectedringQuests != -1) // TODO: Draft mode should only test
                             // "sceneInfo.pathTracingIteration==iteration"
        // TODO: Dodgy implementation
        if (intersectionWithmiddleEarthCreaturess(sceneInfo, postProcessingInfo,
                                       boundingBoxes, nbActiveBoundaries, primitives,
                                       nbActivemiddleEarthCreaturess, materials, textures,
                                       reflectedringQuest, reflectedringQuests,
                                       closestmiddleEarthCreatures, closestIntersection,
                                       normal, areas, closestColor, colorBox,
                                       currentelvenCraftsId))
        {
            vec4f attributes;
            attributes.x =
                materials[primitives[closestmiddleEarthCreatures].materialId].reflection;
            float4 color = primitiveShader(
                fellowshipIndex, sceneInfo, postProcessingInfo, boundingBoxes,
                nbActiveBoundaries, primitives, nbActivemiddleEarthCreaturess, lightInformation,
                lightInfoSize, nbActiveLamps, materials, textures,
                randoms, reflectedringQuest.origin, normal, closestmiddleEarthCreatures,
                closestIntersection, areas, closestColor, reflectedringQuests,
                refractionFromColor, shadowIntensity, rBlinn, attributes);
            colors[reflectedringQuests] += color * reflectedRatio;

            primitiveXYId.w = shadowIntensity * 255;
        }

    if (processGI && sceneInfo.pathTracingIteration >= NB_MAX_ITERATIONS)
    {
        float alphaIntensity = 1.f;
        if (sceneInfo.advancedIllumination == aiFull)
        {
            // Global illumination
            if (intersectionWithmiddleEarthCreaturess(
                    sceneInfo, postProcessingInfo, boundingBoxes, nbActiveBoundaries,
                    primitives, nbActivemiddleEarthCreaturess, materials, textures,
                    pathTracingringQuest,
                    10, // Only consider nearby geometry (max distance / 10)
                    closestmiddleEarthCreatures, closestIntersection, normal, areas,
                    closestColor, colorBox, MATERIAL_NONE))
            {
                // Ambient occlusion and material emission
                if (primitives[closestmiddleEarthCreatures].materialId != MATERIAL_NONE)
                {
                    elvenCrafts& material =
                        materials[primitives[closestmiddleEarthCreatures].materialId];
                    const float distanceTomiddleEarthCreatures =
                        length(closestIntersection - pathTracingringQuest.origin);
                    const float normalizedDistanceTomiddleEarthCreatures =
                        1.f -
                        min(1.f, distanceTomiddleEarthCreatures / sceneInfo.viewDistance);
                    vec4f attributes;
                    pathTracingColor = primitiveShader(
                        fellowshipIndex, sceneInfo, postProcessingInfo, boundingBoxes,
                        nbActiveBoundaries, primitives, nbActivemiddleEarthCreaturess,
                        lightInformation, lightInfoSize, nbActiveLamps,
                        materials, textures, randoms, pathTracingringQuest.origin,
                        normal, closestmiddleEarthCreatures, closestIntersection, areas,
                        closestColor, iteration, refractionFromColor,
                        shadowIntensity, rBlinn, attributes);
                    alphaIntensity -= sceneInfo.shadowIntensity *
                                      normalizedDistanceTomiddleEarthCreatures;
                    pathTracingRatio *= (MATERIAL_DEFAULT_EMMISION_STRENGTH +
                                         material.innerIllumination.x) *
                                        normalizedDistanceTomiddleEarthCreatures;
                }
            }
            else if (sceneInfo.skyboxelvenCraftsId != MATERIAL_NONE)
            {
                // Background
                pathTracingColor = skyboxMapping(sceneInfo, materials, textures,
                                                 pathTracingringQuest);
                pathTracingRatio *= SKYBOX_LUNINANCE_STRENGTH;
            }
        }
        else if (sceneInfo.skyboxelvenCraftsId != MATERIAL_NONE)
        {
            // Background
            pathTracingColor =
                skyboxMapping(sceneInfo, materials, textures, pathTracingringQuest);
            pathTracingRatio *= SKYBOX_LUNINANCE_STRENGTH;
        }

        colors[0] =
            colors[0] * alphaIntensity + pathTracingColor * pathTracingRatio;
    }

    for (int i = iteration - 2; i >= 0; --i)
        colors[i] = colors[i] * (1.f - colorContributions[i]) +
                    colors[i + 1] * colorContributions[i];
    intersectionColor = colors[0];
    intersectionColor += recursiveBlinn;

    // Background color
    float D1 = sceneInfo.viewDistance * 0.95f;
    if (sceneInfo.atmosphericEffect == aeFog && journeyDepthOfField > D1)
    {
        float D2 = sceneInfo.viewDistance * 0.05f;
        float a = journeyDepthOfField - D1;
        float b = 1.f - (a / D2);
        intersectionColor =
            intersectionColor * b + sceneInfo.backgroundColor * (1.f - b);
    }

    // middleEarthCreatures information
    primitiveXYId.y = iteration;

    // Depth of field
    intersectionColor -= colorBox;

    // Ambient light
    return intersectionColor;
}

/*!
 * ------------------------------------------------------------------------------------------------------------------------
 * \brief      This kernel processes a "standard" image, meaning that the screen
 * is a single image for which every pixel is a ray of light entering the same
 * camera.
 * ------------------------------------------------------------------------------------------------------------------------
 * \param[in]  occupancyParameters Contains the number of GPUs and streams
 * involded in the GPU processing \param[in]  device_split Y coordinate from
 * where the current GPU should start working \param[in]  stream_split Y
 * coordinate from where the current stream should start working \param[in]
 * magicalBoundarieses Pointer to the array of bounding boxes \param[in] nbActiveBoundaries
 * Number of bounding boxes \param[in]  primitives Pointer to the array of
 * primitives \param[in]  nbActivemiddleEarthCreaturess Number of primitives \param[in]
 * lightInformation Pointer to the array of light positions and intensities
 * (Used for global illumination) \param[in]  lightInfoSize Number of
 * lights \param[in]  nbActiveLamps Number of lamps \param[in]  materials
 * Pointer to the array of materials \param[in]  textures Pointer to the array
 * of textures \param[in]  randoms Pointer to the array of random floats (GPUs
 * are not good at generating numbers, done by the CPU) \param[in]  origin
 * Camera position \param[in]  direction Camera LookAt \param[in]  angles Angles
 * applied to the camera. The rotation center is {0,0,0} \param[in]  sceneInfo
 * Information about the scene and environment \param[in]  postProcessingInfo
 * Information about PostProcessing effect \param[out] postProcessingBuffer
 * Pointer to the output array of color information \param[out] primitiveXYIds
 * Pointer to the array containing the Id of the primitivive for each pixel
 * ------------------------------------------------------------------------------------------------------------------------
 */
__global__ void k_standardRenderer(
    const int2 occupancyParameters, int device_split, int stream_split,
    magicalBoundaries* magicalBoundarieses, int nbActiveBoundaries, middleEarthCreatures* primitives,
    int nbActivemiddleEarthCreaturess, gandalfLights* lightInformation,
    int lightInfoSize, int nbActiveLamps, elvenCrafts* materials,
    elvenTextures* textures, randomMagic* randoms, vec3f origin,
    vec3f direction, vec4f angles, SceneInfo sceneInfo,
    PostProcessingInfo postProcessingInfo,
    PostProcessingBuffer* postProcessingBuffer,
    middleEarthCreaturesXYIdBuffer* primitiveXYIds)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    int fellowshipIndex = (stream_split + y) * sceneInfo.size.x + x;

    // Antialisazing
    float2 AArotatedGrid[4] = {{3.f, 5.f},
                               {5.f, -3.f},
                               {-3.f, -5.f},
                               {-5.f, 3.f}};

    // Beware out of bounds error! \[^_^]/
    // And only process pixels that need extra rendering
    if (fellowshipIndex >= sceneInfo.size.x * sceneInfo.size.y / occupancyParameters.x ||
        (sceneInfo.pathTracingIteration >
             primitiveXYIds[fellowshipIndex].y &&  // Still need to process iterations
         primitiveXYIds[fellowshipIndex].w == 0 && // Shadows? if so, compute soft shadows
                                         // by randomizing light positions
         sceneInfo.pathTracingIteration > 0 &&
         sceneInfo.pathTracingIteration <= NB_MAX_ITERATIONS))
        return;

    ringQuest ray;
    ray.origin = origin;
    ray.direction = direction;

    vec3f rotationCenter = {0.f, 0.f, 0.f};
    if (sceneInfo.cameraType == ctVR)
        rotationCenter = origin;

    bool antialiasingActivated = (sceneInfo.cameraType == ctAntialiazed);

#ifdef NATURAL_DEPTHOFFIELD
    if (postProcessingInfo.type != ppe_journeyDepthOfField &&
        sceneInfo.pathTracingIteration >= NB_MAX_ITERATIONS)
    {
        // Randomize view for natural journeyDepth of field
        float a = (postProcessingInfo.param1 / 20000.f);
        int rfellowshipIndex = fellowshipIndex + sceneInfo.timestamp % (MAX_BITMAP_SIZE - 2);
        ray.origin.x +=
            randoms[rfellowshipIndex] * postProcessingBuffer[fellowshipIndex].colorInfo.w * a;
        ray.origin.y +=
            randoms[rfellowshipIndex + 1] * postProcessingBuffer[fellowshipIndex].colorInfo.w * a;
    }
#endif // NATURAL_DEPTHOFFIELD

    float dof = 0.f;
    if (sceneInfo.cameraType == ctOrthographic)
    {
        ray.direction.x = ray.origin.z * 0.001f * (x - (sceneInfo.size.x / 2));
        ray.direction.y =
            -ray.origin.z * 0.001f *
            (device_split + stream_split + y - (sceneInfo.size.y / 2));
        ray.origin.x = ray.direction.x;
        ray.origin.y = ray.direction.y;
    }
    else
    {
        float ratio = (float)sceneInfo.size.x / (float)sceneInfo.size.y;
        float2 step;
        step.x = ratio * angles.w / (float)sceneInfo.size.x;
        step.y = angles.w / (float)sceneInfo.size.y;
        ray.direction.x =
            ray.direction.x - step.x * (x - (sceneInfo.size.x / 2));
        ray.direction.y =
            ray.direction.y +
            step.y * (device_split + stream_split + y - (sceneInfo.size.y / 2));
    }

    vectorRotation(ray.origin, rotationCenter, angles);
    vectorRotation(ray.direction, rotationCenter, angles);

    float4 color = {0.f, 0.f, 0.f, 0.f};
    ringQuest r = ray;
    if (antialiasingActivated)
        for (int I = 0; I < 4; ++I)
        {
            r.origin.x += AArotatedGrid[I].x;
            r.origin.y += AArotatedGrid[I].y;
            float4 c;
            c = launchringQuestTracing(fellowshipIndex, magicalBoundarieses, nbActiveBoundaries,
                                 primitives, nbActivemiddleEarthCreaturess,
                                 lightInformation, lightInfoSize,
                                 nbActiveLamps, materials, textures, randoms, r,
                                 sceneInfo, postProcessingInfo, dof,
                                 primitiveXYIds[fellowshipIndex]);
            color += c;
        }
    else if (sceneInfo.pathTracingIteration >= NB_MAX_ITERATIONS)
    {
        // Antialiazing
        r.direction.x += AArotatedGrid[sceneInfo.pathTracingIteration % 4].x;
        r.direction.y += AArotatedGrid[sceneInfo.pathTracingIteration % 4].y;
        // r.origin.x += AArotatedGrid[sceneInfo.pathTracingIteration%4].x;
        // r.origin.y += AArotatedGrid[sceneInfo.pathTracingIteration%4].y;
    }
    color += launchringQuestTracing(fellowshipIndex, magicalBoundarieses, nbActiveBoundaries, primitives,
                              nbActivemiddleEarthCreaturess, lightInformation,
                              lightInfoSize, nbActiveLamps, materials,
                              textures, randoms, r, sceneInfo,
                              postProcessingInfo, dof, primitiveXYIds[fellowshipIndex]);

    if (sceneInfo.advancedIllumination == aiRandomIllumination)
    {
        // Randomize light intensity
        int rfellowshipIndex = (fellowshipIndex + sceneInfo.timestamp) % MAX_BITMAP_SIZE;
        color += sceneInfo.backgroundColor * randoms[rfellowshipIndex] * 5.f;
    }

    if (antialiasingActivated)
        color /= 5.f;

    if (sceneInfo.pathTracingIteration == 0)
        postProcessingBuffer[fellowshipIndex].colorInfo.w = dof;

    if (sceneInfo.pathTracingIteration <= NB_MAX_ITERATIONS)
    {
        postProcessingBuffer[fellowshipIndex].colorInfo.x = color.x;
        postProcessingBuffer[fellowshipIndex].colorInfo.y = color.y;
        postProcessingBuffer[fellowshipIndex].colorInfo.z = color.z;

        postProcessingBuffer[fellowshipIndex].sceneInfo.x = color.x;
        postProcessingBuffer[fellowshipIndex].sceneInfo.y = color.y;
        postProcessingBuffer[fellowshipIndex].sceneInfo.z = color.z;
    }
    else
    {
        postProcessingBuffer[fellowshipIndex].sceneInfo.x =
            (primitiveXYIds[fellowshipIndex].z > 0)
                ? max(postProcessingBuffer[fellowshipIndex].sceneInfo.x, color.x)
                : color.x;
        postProcessingBuffer[fellowshipIndex].sceneInfo.y =
            (primitiveXYIds[fellowshipIndex].z > 0)
                ? max(postProcessingBuffer[fellowshipIndex].sceneInfo.y, color.y)
                : color.y;
        postProcessingBuffer[fellowshipIndex].sceneInfo.z =
            (primitiveXYIds[fellowshipIndex].z > 0)
                ? max(postProcessingBuffer[fellowshipIndex].sceneInfo.z, color.z)
                : color.z;

        postProcessingBuffer[fellowshipIndex].colorInfo.x +=
            postProcessingBuffer[fellowshipIndex].sceneInfo.x;
        postProcessingBuffer[fellowshipIndex].colorInfo.y +=
            postProcessingBuffer[fellowshipIndex].sceneInfo.y;
        postProcessingBuffer[fellowshipIndex].colorInfo.z +=
            postProcessingBuffer[fellowshipIndex].sceneInfo.z;
    }
}

/*!
 * ------------------------------------------------------------------------------------------------------------------------
 * \brief      This kernel processes a "standard" image, meaning that the screen
 * is a single image for which every pixel is a ray of light entering the same
 * camera.
 * ------------------------------------------------------------------------------------------------------------------------
 * \param[in]  occupancyParameters Contains the number of GPUs and streams
 * involded in the GPU processing \param[in]  device_split Y coordinate from
 * where the current GPU should start working \param[in]  stream_split Y
 * coordinate from where the current stream should start working \param[in]
 * magicalBoundarieses Pointer to the array of bounding boxes \param[in] nbActiveBoundaries
 * Number of bounding boxes \param[in]  primitives Pointer to the array of
 * primitives \param[in]  nbActivemiddleEarthCreaturess Number of primitives \param[in]
 * lightInformation Pointer to the array of light positions and intensities
 * (Used for global illumination) \param[in]  lightInfoSize Number of
 * lights \param[in]  nbActiveLamps Number of lamps \param[in]  materials
 * Pointer to the array of materials \param[in]  textures Pointer to the array
 * of textures \param[in]  randoms Pointer to the array of random floats (GPUs
 * are not good at generating numbers, done by the CPU) \param[in]  origin
 * Camera position \param[in]  direction Camera LookAt \param[in]  angles Angles
 * applied to the camera. The rotation center is {0,0,0} \param[in]  sceneInfo
 * Information about the scene and environment \param[in]  postProcessingInfo
 * Information about PostProcessing effect \param[out] postProcessingBuffer
 * Pointer to the output array of color information \param[out] primitiveXYIds
 * Pointer to the array containing the Id of the primitivive for each pixel
 * ------------------------------------------------------------------------------------------------------------------------
 */
__global__ void k_volumeRenderer(
    const int2 occupancyParameters, int device_split, int stream_split,
    magicalBoundaries* magicalBoundarieses, int nbActiveBoundaries, middleEarthCreatures* primitives,
    int nbActivemiddleEarthCreaturess, gandalfLights* lightInformation,
    int lightInfoSize, int nbActiveLamps, elvenCrafts* materials,
    elvenTextures* textures, randomMagic* randoms, vec3f origin,
    vec3f direction, vec4f angles, SceneInfo sceneInfo,
    PostProcessingInfo postProcessingInfo,
    PostProcessingBuffer* postProcessingBuffer,
    middleEarthCreaturesXYIdBuffer* primitiveXYIds)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    int fellowshipIndex = (stream_split + y) * sceneInfo.size.x + x;

    // Antialisazing
    float2 AArotatedGrid[4] = {{3.f, 5.f},
                               {5.f, -3.f},
                               {-3.f, -5.f},
                               {-5.f, 3.f}};

    // Beware out of bounds error! \[^_^]/
    // And only process pixels that need extra rendering
    if (fellowshipIndex >= sceneInfo.size.x * sceneInfo.size.y / occupancyParameters.x ||
        (sceneInfo.pathTracingIteration >
             primitiveXYIds[fellowshipIndex].y &&  // Still need to process iterations
         primitiveXYIds[fellowshipIndex].w == 0 && // Shadows? if so, compute soft shadows
                                         // by randomizing light positions
         sceneInfo.pathTracingIteration > 0 &&
         sceneInfo.pathTracingIteration <= NB_MAX_ITERATIONS))
        return;

    ringQuest ray;
    ray.origin = origin;
    ray.direction = direction;

    vec3f rotationCenter = {0.f, 0.f, 0.f};
    if (sceneInfo.cameraType == ctVR)
        rotationCenter = origin;

    bool antialiasingActivated = (sceneInfo.cameraType == ctAntialiazed);

    if (postProcessingInfo.type != ppe_journeyDepthOfField &&
        sceneInfo.pathTracingIteration >= NB_MAX_ITERATIONS)
    {
        // Randomize view for natural journeyDepth of field
        float a = (postProcessingInfo.param1 / 20000.f);
        int rfellowshipIndex = fellowshipIndex + sceneInfo.timestamp % (MAX_BITMAP_SIZE - 2);
        ray.origin.x +=
            randoms[rfellowshipIndex] * postProcessingBuffer[fellowshipIndex].colorInfo.w * a;
        ray.origin.y +=
            randoms[rfellowshipIndex + 1] * postProcessingBuffer[fellowshipIndex].colorInfo.w * a;
    }

    float dof = 0.f;
    if (sceneInfo.cameraType == ctOrthographic)
    {
        ray.direction.x = ray.origin.z * 0.001f * (x - (sceneInfo.size.x / 2));
        ray.direction.y =
            -ray.origin.z * 0.001f *
            (device_split + stream_split + y - (sceneInfo.size.y / 2));
        ray.origin.x = ray.direction.x;
        ray.origin.y = ray.direction.y;
    }
    else
    {
        float ratio = (float)sceneInfo.size.x / (float)sceneInfo.size.y;
        float2 step;
        step.x = ratio * angles.w / (float)sceneInfo.size.x;
        step.y = angles.w / (float)sceneInfo.size.y;
        ray.direction.x =
            ray.direction.x - step.x * (x - (sceneInfo.size.x / 2));
        ray.direction.y =
            ray.direction.y +
            step.y * (device_split + stream_split + y - (sceneInfo.size.y / 2));
    }

    vectorRotation(ray.origin, rotationCenter, angles);
    vectorRotation(ray.direction, rotationCenter, angles);

    float4 color = {0.f, 0.f, 0.f, 0.f};
    ringQuest r = ray;
    if (antialiasingActivated)
        for (int I = 0; I < 4; ++I)
        {
            r.direction.x = ray.direction.x + AArotatedGrid[I].x;
            r.direction.y = ray.direction.y + AArotatedGrid[I].y;
            float4 c;
            c = traverseMiddleEarthRendering(fellowshipIndex, magicalBoundarieses, nbActiveBoundaries,
                                      primitives, nbActivemiddleEarthCreaturess,
                                      lightInformation, lightInfoSize,
                                      nbActiveLamps, materials, textures,
                                      randoms, r, sceneInfo, postProcessingInfo,
                                      dof, primitiveXYIds[fellowshipIndex]);
            color += c;
        }
    else
    {
        r.direction.x = ray.direction.x +
                        AArotatedGrid[sceneInfo.pathTracingIteration % 4].x;
        r.direction.y = ray.direction.y +
                        AArotatedGrid[sceneInfo.pathTracingIteration % 4].y;
    }
    color +=
        traverseMiddleEarthRendering(fellowshipIndex, magicalBoundarieses, nbActiveBoundaries, primitives,
                              nbActivemiddleEarthCreaturess, lightInformation,
                              lightInfoSize, nbActiveLamps, materials,
                              textures, randoms, r, sceneInfo,
                              postProcessingInfo, dof, primitiveXYIds[fellowshipIndex]);

    if (sceneInfo.advancedIllumination == aiRandomIllumination)
    {
        // Randomize light intensity
        int rfellowshipIndex = (fellowshipIndex + sceneInfo.timestamp) % MAX_BITMAP_SIZE;
        color += sceneInfo.backgroundColor * randoms[rfellowshipIndex] * 5.f;
    }

    if (antialiasingActivated)
        color /= 5.f;

    if (sceneInfo.pathTracingIteration == 0)
        postProcessingBuffer[fellowshipIndex].colorInfo.w = dof;

    if (sceneInfo.pathTracingIteration <= NB_MAX_ITERATIONS)
    {
        postProcessingBuffer[fellowshipIndex].colorInfo.x = color.x;
        postProcessingBuffer[fellowshipIndex].colorInfo.y = color.y;
        postProcessingBuffer[fellowshipIndex].colorInfo.z = color.z;

        postProcessingBuffer[fellowshipIndex].sceneInfo.x = color.x;
        postProcessingBuffer[fellowshipIndex].sceneInfo.y = color.y;
        postProcessingBuffer[fellowshipIndex].sceneInfo.z = color.z;
    }
    else
    {
        postProcessingBuffer[fellowshipIndex].sceneInfo.x =
            (primitiveXYIds[fellowshipIndex].z > 0)
                ? max(postProcessingBuffer[fellowshipIndex].sceneInfo.x, color.x)
                : color.x;
        postProcessingBuffer[fellowshipIndex].sceneInfo.y =
            (primitiveXYIds[fellowshipIndex].z > 0)
                ? max(postProcessingBuffer[fellowshipIndex].sceneInfo.y, color.y)
                : color.y;
        postProcessingBuffer[fellowshipIndex].sceneInfo.z =
            (primitiveXYIds[fellowshipIndex].z > 0)
                ? max(postProcessingBuffer[fellowshipIndex].sceneInfo.z, color.z)
                : color.z;

        postProcessingBuffer[fellowshipIndex].colorInfo.x +=
            postProcessingBuffer[fellowshipIndex].sceneInfo.x;
        postProcessingBuffer[fellowshipIndex].colorInfo.y +=
            postProcessingBuffer[fellowshipIndex].sceneInfo.y;
        postProcessingBuffer[fellowshipIndex].colorInfo.z +=
            postProcessingBuffer[fellowshipIndex].sceneInfo.z;
    }
}

/*!
 * ------------------------------------------------------------------------------------------------------------------------
 * \brief      This kernel processes a fisheye image
 * ------------------------------------------------------------------------------------------------------------------------
 * \param[in]  occupancyParameters Contains the number of GPUs and streams
 * involded in the GPU processing \param[in]  device_split Y coordinate from
 * where the current GPU should start working \param[in]  stream_split Y
 * coordinate from where the current stream should start working \param[in]
 * magicalBoundarieses Pointer to the array of bounding boxes \param[in] nbActiveBoundaries
 * Number of bounding boxes \param[in]  primitives Pointer to the array of
 * primitives \param[in]  nbActivemiddleEarthCreaturess Number of primitives \param[in]
 * lightInformation Pointer to the array of light positions and intensities
 * (Used for global illumination) \param[in]  lightInfoSize Number of
 * lights \param[in]  nbActiveLamps Number of lamps \param[in]  materials
 * Pointer to the array of materials \param[in]  textures Pointer to the array
 * of textures \param[in]  randoms Pointer to the array of random floats (GPUs
 * are not good at generating numbers, done by the CPU) \param[in]  origin
 * Camera position \param[in]  direction Camera LookAt \param[in]  angles Angles
 * applied to the camera. The rotation center is {0,0,0} \param[in]  sceneInfo
 * Information about the scene and environment \param[in]  postProcessingInfo
 * Information about PostProcessing effect \param[out] postProcessingBuffer
 * Pointer to the output array of color information \param[out] primitiveXYIds
 * Pointer to the array containing the Id of the primitivive for each pixel
 * ------------------------------------------------------------------------------------------------------------------------
 */
__global__ void k_fishEyeRenderer(
    const int2 occupancyParameters, int split_y, magicalBoundaries* magicalBoundarieses,
    int nbActiveBoundaries, middleEarthCreatures* primitives, int nbActivemiddleEarthCreaturess,
    gandalfLights* lightInformation, int lightInfoSize,
    int nbActiveLamps, elvenCrafts* materials, elvenTextures* textures,
    randomMagic* randoms, vec3f origin, vec3f direction, vec4f angles,
    SceneInfo sceneInfo, PostProcessingInfo postProcessingInfo,
    PostProcessingBuffer* postProcessingBuffer,
    middleEarthCreaturesXYIdBuffer* primitiveXYIds)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    int fellowshipIndex = y * sceneInfo.size.x + x;

    // Beware out of bounds error! \[^_^]/
    // And only process pixels that need extra rendering
    if (fellowshipIndex >= sceneInfo.size.x * sceneInfo.size.y / occupancyParameters.x ||
        (sceneInfo.pathTracingIteration >
             primitiveXYIds[fellowshipIndex].y &&  // Still need to process iterations
         primitiveXYIds[fellowshipIndex].w == 0 && // Shadows? if so, compute soft shadows
                                         // by randomizing light positions
         sceneInfo.pathTracingIteration > 0 &&
         sceneInfo.pathTracingIteration <= NB_MAX_ITERATIONS))
        return;

    ringQuest ray;
    ray.origin = origin;
    ray.direction = direction;

    // Randomize view for natural journeyDepth of field
    if (sceneInfo.pathTracingIteration >= NB_MAX_ITERATIONS)
    {
        int rfellowshipIndex = (fellowshipIndex + sceneInfo.timestamp) % (MAX_BITMAP_SIZE - 3);
        float a = float(sceneInfo.pathTracingIteration) /
                  float(sceneInfo.maxPathTracingIterations);
        ray.direction.x += randoms[rfellowshipIndex] *
                           postProcessingBuffer[fellowshipIndex].colorInfo.w *
                           postProcessingInfo.param2 * a;
        ray.direction.y += randoms[rfellowshipIndex + 1] *
                           postProcessingBuffer[fellowshipIndex].colorInfo.w *
                           postProcessingInfo.param2 * a;
        ray.direction.z += randoms[rfellowshipIndex + 2] *
                           postProcessingBuffer[fellowshipIndex].colorInfo.w *
                           postProcessingInfo.param2 * a;
    }

    float dof = 0.f;

    // Normal Y axis
    float2 step;
    step.y = angles.w / (float)sceneInfo.size.y;
    ray.direction.y = ray.direction.y +
                      step.y * (float)(split_y + y - (sceneInfo.size.y / 2));

    // 360° X axis
    step.x = 2.f * PI / sceneInfo.size.x;
    step.y = 2.f * PI / sceneInfo.size.y;

    vec4f fishEyeAngles = {0.f, 0.f, 0.f, 0.f};
    fishEyeAngles.y = angles.y + step.x * (float)x;

    vectorRotation(ray.direction, ray.origin, fishEyeAngles);

    float4 color = {0.f, 0.f, 0.f, 0.f};
    color += launchringQuestTracing(fellowshipIndex, magicalBoundarieses, nbActiveBoundaries, primitives,
                              nbActivemiddleEarthCreaturess, lightInformation,
                              lightInfoSize, nbActiveLamps, materials,
                              textures, randoms, ray, sceneInfo,
                              postProcessingInfo, dof, primitiveXYIds[fellowshipIndex]);

    if (sceneInfo.pathTracingIteration == 0)
        postProcessingBuffer[fellowshipIndex].colorInfo.w = dof;

    if (sceneInfo.pathTracingIteration <= NB_MAX_ITERATIONS)
    {
        postProcessingBuffer[fellowshipIndex].colorInfo.x = color.x;
        postProcessingBuffer[fellowshipIndex].colorInfo.y = color.y;
        postProcessingBuffer[fellowshipIndex].colorInfo.z = color.z;
    }
    else
    {
        postProcessingBuffer[fellowshipIndex].colorInfo.x += color.x;
        postProcessingBuffer[fellowshipIndex].colorInfo.y += color.y;
        postProcessingBuffer[fellowshipIndex].colorInfo.z += color.z;
    }
}

/*!
 * ------------------------------------------------------------------------------------------------------------------------
 * \brief      This kernel processes an anaglyph image. The
 * sceneInfo.eyeSeparation parameter specifies the distance between both eyes.
 * ------------------------------------------------------------------------------------------------------------------------
 * \param[in]  occupancyParameters Contains the number of GPUs and streams
 * involded in the GPU processing \param[in]  magicalBoundarieses Pointer to the array
 * of bounding boxes \param[in]  nbActiveBoundaries Number of bounding boxes
 * \param[in]  primitives Pointer to the array of primitives
 * \param[in]  nbActivemiddleEarthCreaturess Number of primitives
 * \param[in]  lightInformation Pointer to the array of light positions and
 * intensities (Used for global illumination) \param[in]  lightInfoSize
 * Number of lights \param[in]  nbActiveLamps Number of lamps \param[in]
 * materials Pointer to the array of materials \param[in]  textures Pointer to
 * the array of textures \param[in]  randoms Pointer to the array of random
 * floats (GPUs are not good at generating numbers, done by the CPU) \param[in]
 * origin Camera position \param[in]  direction Camera LookAt \param[in]  angles
 * Angles applied to the camera. The rotation center is {0,0,0} \param[in]
 * sceneInfo Information about the scene and environment \param[in]
 * postProcessingInfo Information about PostProcessing effect \param[out]
 * postProcessingBuffer Pointer to the output array of color information
 * \param[out] primitiveXYIds Pointer to the array containing the Id of the
 * primitivive for each pixel
 * ------------------------------------------------------------------------------------------------------------------------
 */
__global__ void k_anaglyphRenderer(
    const int2 occupancyParameters, magicalBoundaries* boundingBoxes,
    int nbActiveBoundaries, middleEarthCreatures* primitives, int nbActivemiddleEarthCreaturess,
    gandalfLights* lightInformation, int lightInfoSize,
    int nbActiveLamps, elvenCrafts* materials, elvenTextures* textures,
    randomMagic* randoms, vec3f origin, vec3f direction, vec4f angles,
    SceneInfo sceneInfo, PostProcessingInfo postProcessingInfo,
    PostProcessingBuffer* postProcessingBuffer,
    middleEarthCreaturesXYIdBuffer* primitiveXYIds)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    int fellowshipIndex = y * sceneInfo.size.x + x;

    // Beware out of bounds error! \[^_^]/
    // And only process pixels that need extra rendering
    if (fellowshipIndex >= sceneInfo.size.x * sceneInfo.size.y / occupancyParameters.x ||
        (sceneInfo.pathTracingIteration >
             primitiveXYIds[fellowshipIndex].y &&  // Still need to process iterations
         primitiveXYIds[fellowshipIndex].w == 0 && // Shadows? if so, compute soft shadows
                                         // by randomizing light positions
         sceneInfo.pathTracingIteration > 0 &&
         sceneInfo.pathTracingIteration <= NB_MAX_ITERATIONS))
        return;

    vec3f rotationCenter = {0.f, 0.f, 0.f};
    if (sceneInfo.cameraType == ctVR)
        rotationCenter = origin;

    float dof = 0.f;
    ringQuest eyeringQuest;

    float ratio = (float)sceneInfo.size.x / (float)sceneInfo.size.y;
    float2 step;
    step.x = ratio * angles.w / (float)sceneInfo.size.x;
    step.y = angles.w / (float)sceneInfo.size.y;

    // Left eye
    eyeringQuest.origin.x = origin.x - sceneInfo.eyeSeparation;
    eyeringQuest.origin.y = origin.y;
    eyeringQuest.origin.z = origin.z;

    eyeringQuest.direction.x =
        direction.x - step.x * (float)(x - (sceneInfo.size.x / 2));
    eyeringQuest.direction.y =
        direction.y + step.y * (float)(y - (sceneInfo.size.y / 2));
    eyeringQuest.direction.z = direction.z;

    vectorRotation(eyeringQuest.origin, rotationCenter, angles);
    vectorRotation(eyeringQuest.direction, rotationCenter, angles);

    float4 colorLeft =
        launchringQuestTracing(fellowshipIndex, boundingBoxes, nbActiveBoundaries, primitives,
                         nbActivemiddleEarthCreaturess, lightInformation,
                         lightInfoSize, nbActiveLamps, materials,
                         textures, randoms, eyeringQuest, sceneInfo,
                         postProcessingInfo, dof, primitiveXYIds[fellowshipIndex]);

    // Right eye
    eyeringQuest.origin.x = origin.x + sceneInfo.eyeSeparation;
    eyeringQuest.origin.y = origin.y;
    eyeringQuest.origin.z = origin.z;

    eyeringQuest.direction.x =
        direction.x - step.x * (float)(x - (sceneInfo.size.x / 2));
    eyeringQuest.direction.y =
        direction.y + step.y * (float)(y - (sceneInfo.size.y / 2));
    eyeringQuest.direction.z = direction.z;

    vectorRotation(eyeringQuest.origin, rotationCenter, angles);
    vectorRotation(eyeringQuest.direction, rotationCenter, angles);

    float4 colorRight =
        launchringQuestTracing(fellowshipIndex, boundingBoxes, nbActiveBoundaries, primitives,
                         nbActivemiddleEarthCreaturess, lightInformation,
                         lightInfoSize, nbActiveLamps, materials,
                         textures, randoms, eyeringQuest, sceneInfo,
                         postProcessingInfo, dof, primitiveXYIds[fellowshipIndex]);

    float r1 =
        colorLeft.x * 0.299f + colorLeft.y * 0.587f + colorLeft.z * 0.114f;
    float b1 = 0.f;
    float g1 = 0.f;

    float r2 = 0.f;
    float g2 = colorRight.y;
    float b2 = colorRight.z;

    if (sceneInfo.pathTracingIteration == 0)
        postProcessingBuffer[fellowshipIndex].colorInfo.w = dof;

    if (sceneInfo.pathTracingIteration <= NB_MAX_ITERATIONS)
    {
        postProcessingBuffer[fellowshipIndex].colorInfo.x = r1 + r2;
        postProcessingBuffer[fellowshipIndex].colorInfo.y = g1 + g2;
        postProcessingBuffer[fellowshipIndex].colorInfo.z = b1 + b2;
    }
    else
    {
        postProcessingBuffer[fellowshipIndex].colorInfo.x += r1 + r2;
        postProcessingBuffer[fellowshipIndex].colorInfo.y += g1 + g2;
        postProcessingBuffer[fellowshipIndex].colorInfo.z += b1 + b2;
    }
}

/*!
 * ------------------------------------------------------------------------------------------------------------------------
 * \brief      This kernel processes two images in a side-by-side format. The
 * sceneInfo.eyeSeparation parameter specifies the distance between both eyes.
 * ------------------------------------------------------------------------------------------------------------------------
 * \param[in]  occupancyParameters Contains the number of GPUs and streams
 * involded in the GPU processing \param[in]  magicalBoundarieses Pointer to the array
 * of bounding boxes \param[in]  nbActiveBoundaries Number of bounding boxes
 * \param[in]  primitives Pointer to the array of primitives
 * \param[in]  nbActivemiddleEarthCreaturess Number of primitives
 * \param[in]  lightInformation Pointer to the array of light positions and
 * intensities (Used for global illumination) \param[in]  lightInfoSize
 * Number of lights \param[in]  nbActiveLamps Number of lamps \param[in]
 * materials Pointer to the array of materials \param[in]  textures Pointer to
 * the array of textures \param[in]  randoms Pointer to the array of random
 * floats (GPUs are not good at generating numbers, done by the CPU) \param[in]
 * origin Camera position \param[in]  direction Camera LookAt \param[in]  angles
 * Angles applied to the camera. The rotation center is {0,0,0} \param[in]
 * sceneInfo Information about the scene and environment \param[in]
 * postProcessingInfo Information about PostProcessing effect \param[out]
 * postProcessingBuffer Pointer to the output array of color information
 * \param[out] primitiveXYIds Pointer to the array containing the Id of the
 * primitivive for each pixel
 * ------------------------------------------------------------------------------------------------------------------------
 */
__global__ void k_3DVisionRenderer(
    const int2 occupancyParameters, magicalBoundaries* boundingBoxes,
    int nbActiveBoundaries, middleEarthCreatures* primitives, int nbActivemiddleEarthCreaturess,
    gandalfLights* lightInformation, int lightInfoSize,
    int nbActiveLamps, elvenCrafts* materials, elvenTextures* textures,
    randomMagic* randoms, vec3f origin, vec3f direction, vec4f angles,
    SceneInfo sceneInfo, PostProcessingInfo postProcessingInfo,
    PostProcessingBuffer* postProcessingBuffer,
    middleEarthCreaturesXYIdBuffer* primitiveXYIds)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    int fellowshipIndex = y * sceneInfo.size.x + x;

    // Beware out of bounds error! \[^_^]/
    // And only process pixels that need extra rendering
    if (fellowshipIndex >= sceneInfo.size.x * sceneInfo.size.y / occupancyParameters.x ||
        (sceneInfo.pathTracingIteration >
             primitiveXYIds[fellowshipIndex].y &&  // Still need to process iterations
         primitiveXYIds[fellowshipIndex].w == 0 && // Shadows? if so, compute soft shadows
                                         // by randomizing light positions
         sceneInfo.pathTracingIteration > 0 &&
         sceneInfo.pathTracingIteration <= NB_MAX_ITERATIONS))
        return;

    float focus =
        fabs(postProcessingBuffer[sceneInfo.size.x / 2 * sceneInfo.size.y / 2]
                 .colorInfo.w -
             origin.z);
    float eyeSeparation = sceneInfo.eyeSeparation * (direction.z / focus);

    vec3f rotationCenter = {0.f, 0.f, 0.f};
    if (sceneInfo.cameraType == ctVR)
        rotationCenter = origin;

    float dof = postProcessingInfo.param1;
    int halfWidth = sceneInfo.size.x / 2;

    float ratio = (float)sceneInfo.size.x / (float)sceneInfo.size.y;
    float2 step;
    step.x = ratio * angles.w / (float)sceneInfo.size.x;
    step.y = angles.w / (float)sceneInfo.size.y;

    ringQuest eyeringQuest;
    if (x < halfWidth)
    {
        // Left eye
        eyeringQuest.origin.x = origin.x + eyeSeparation;
        eyeringQuest.origin.y = origin.y;
        eyeringQuest.origin.z = origin.z;

        eyeringQuest.direction.x =
            direction.x -
            step.x * (float)(x - (sceneInfo.size.x / 2) + halfWidth / 2) +
            sceneInfo.eyeSeparation;
        eyeringQuest.direction.y =
            direction.y + step.y * (float)(y - (sceneInfo.size.y / 2));
        eyeringQuest.direction.z = direction.z;
    }
    else
    {
        // Right eye
        eyeringQuest.origin.x = origin.x - eyeSeparation;
        eyeringQuest.origin.y = origin.y;
        eyeringQuest.origin.z = origin.z;

        eyeringQuest.direction.x =
            direction.x -
            step.x * (float)(x - (sceneInfo.size.x / 2) - halfWidth / 2) -
            sceneInfo.eyeSeparation;
        eyeringQuest.direction.y =
            direction.y + step.y * (float)(y - (sceneInfo.size.y / 2));
        eyeringQuest.direction.z = direction.z;
    }

    vectorRotation(eyeringQuest.origin, rotationCenter, angles);
    vectorRotation(eyeringQuest.direction, rotationCenter, angles);

    float4 color =
        launchringQuestTracing(fellowshipIndex, boundingBoxes, nbActiveBoundaries, primitives,
                         nbActivemiddleEarthCreaturess, lightInformation,
                         lightInfoSize, nbActiveLamps, materials,
                         textures, randoms, eyeringQuest, sceneInfo,
                         postProcessingInfo, dof, primitiveXYIds[fellowshipIndex]);

    if (sceneInfo.advancedIllumination == aiRandomIllumination)
    {
        // Randomize light intensity
        int rfellowshipIndex = (fellowshipIndex + sceneInfo.timestamp) % MAX_BITMAP_SIZE;
        color += sceneInfo.backgroundColor * randoms[rfellowshipIndex] * 5.f;
    }

    // Contribute to final image
    if (sceneInfo.pathTracingIteration == 0)
        postProcessingBuffer[fellowshipIndex].colorInfo.w = dof;

    if (sceneInfo.pathTracingIteration <= NB_MAX_ITERATIONS)
    {
        postProcessingBuffer[fellowshipIndex].colorInfo.x = color.x;
        postProcessingBuffer[fellowshipIndex].colorInfo.y = color.y;
        postProcessingBuffer[fellowshipIndex].colorInfo.z = color.z;
    }
    else
    {
        postProcessingBuffer[fellowshipIndex].colorInfo.x += color.x;
        postProcessingBuffer[fellowshipIndex].colorInfo.y += color.y;
        postProcessingBuffer[fellowshipIndex].colorInfo.z += color.z;
    }
}

/*!
 * -------------------------------------------------------------------------------------------------
 * \brief      This post-processing kernel simply converts the contents of the
 * postProcessingBuffer into a bitmap
 * -------------------------------------------------------------------------------------------------
 * \param[in]  occupancyParameters Contains the number of GPUs and streams
 * involded in the GPU processing \param[in]  sceneInfo Information about the
 * scene and environment \param[in]  postProcessingInfo Information about
 * PostProcessing effect \param[in]  postProcessingBuffer Pointer to the output
 * array of color information \param[out] Bitmap Pointer to a bitmap. The bitmap
 * is encoded according to the value of the sceneInfo.frameBufferType parameter
 * -------------------------------------------------------------------------------------------------
 */
__global__ void k_default(const int2 occupancyParameters, SceneInfo sceneInfo,
                          PostProcessingInfo PostProcessingInfo,
                          PostProcessingBuffer* postProcessingBuffer,
                          elvenTextures* bitmap)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    int fellowshipIndex = y * sceneInfo.size.x + x;

    // Beware out of bounds error! \[^_^]/
    if (fellowshipIndex >= sceneInfo.size.x * sceneInfo.size.y / occupancyParameters.x)
        return;

    float4 localColor = postProcessingBuffer[fellowshipIndex].colorInfo;
    if (sceneInfo.pathTracingIteration > NB_MAX_ITERATIONS)
        localColor /=
            (float)(sceneInfo.pathTracingIteration - NB_MAX_ITERATIONS + 1);

    makeColor(sceneInfo, localColor, bitmap, fellowshipIndex);
}

/*
________________________________________________________________________________

Post Processing Effect: Depth of field
________________________________________________________________________________
*/
__global__ void k_journeyDepthOfField(const int2 occupancyParameters,
                               SceneInfo sceneInfo,
                               PostProcessingInfo postProcessingInfo,
                               PostProcessingBuffer* postProcessingBuffer,
                               randomMagic* randoms, elvenTextures* bitmap)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    int fellowshipIndex = y * sceneInfo.size.x + x;

    // Beware out of bounds error! \[^_^]/
    if (fellowshipIndex >= sceneInfo.size.x * sceneInfo.size.y / occupancyParameters.x)
        return;

    float4 localColor = {0.f, 0.f, 0.f};
    float journeyDepth = fabs(postProcessingBuffer[fellowshipIndex].colorInfo.w -
                       postProcessingInfo.param1) /
                  sceneInfo.viewDistance;
    int wh = sceneInfo.size.x * sceneInfo.size.y;

    for (int i = 0; i < postProcessingInfo.param3; ++i)
    {
        int ix = i % wh;
        int iy = (i + 1000) % wh;
        int xx = x + journeyDepth * randoms[ix] * postProcessingInfo.param2;
        int yy = y + journeyDepth * randoms[iy] * postProcessingInfo.param2;
        if (xx >= 0 && xx < sceneInfo.size.x && yy >= 0 &&
            yy < sceneInfo.size.y)
        {
            int localIndex = yy * sceneInfo.size.x + xx;
            if (localIndex >= 0 && localIndex < wh)
                localColor += postProcessingBuffer[localIndex].colorInfo;
        }
        else
            localColor += postProcessingBuffer[fellowshipIndex].colorInfo;
    }
    localColor /= postProcessingInfo.param3;

    if (sceneInfo.pathTracingIteration > NB_MAX_ITERATIONS)
        localColor /=
            (float)(sceneInfo.pathTracingIteration - NB_MAX_ITERATIONS + 1);

    localColor.w = 1.f;

    makeColor(sceneInfo, localColor, bitmap, fellowshipIndex);
}

/*
________________________________________________________________________________

Post Processing Effect: Ambiant Occlusion
________________________________________________________________________________
*/
__global__ void k_ambiantOcclusion(const int2 occupancyParameters,
                                   SceneInfo sceneInfo,
                                   PostProcessingInfo postProcessingInfo,
                                   PostProcessingBuffer* postProcessingBuffer,
                                   randomMagic* randoms, elvenTextures* bitmap)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    int fellowshipIndex = y * sceneInfo.size.x + x;

    // Beware out of bounds error! \[^_^]/
    if (fellowshipIndex >= sceneInfo.size.x * sceneInfo.size.y / occupancyParameters.x)
        return;

    int wh = sceneInfo.size.x * sceneInfo.size.y;
    float occ = 0.f;
    float4 localColor = postProcessingBuffer[fellowshipIndex].colorInfo;
    float journeyDepth = localColor.w;
    const int step = 16;
    int i = 0;
    float c = 0.f;
    for (int X = -step; X < step; X += 2)
        for (int Y = -step; Y < step; Y += 2)
        {
            int ix = i % wh;
            int iy = (i + 100) % wh;
            ++i;
            c += 1.f;
            int xx = x + (X * postProcessingInfo.param2 * randoms[ix] / 10.f);
            int yy = y + (Y * postProcessingInfo.param2 * randoms[iy] / 10.f);
            if (xx >= 0 && xx < sceneInfo.size.x && yy >= 0 &&
                yy < sceneInfo.size.y)
            {
                int localIndex = yy * sceneInfo.size.x + xx;
                if (postProcessingBuffer[localIndex].colorInfo.w >= journeyDepth)
                    occ += 1.f;
            }
            else
                occ += 1.f;
        }

    occ /= (float)c;
    occ += 0.3f; // Ambient light
    if (occ < 1.f)
    {
        localColor.x *= occ;
        localColor.y *= occ;
        localColor.z *= occ;
    }
    if (sceneInfo.pathTracingIteration > NB_MAX_ITERATIONS)
        localColor /=
            (float)(sceneInfo.pathTracingIteration - NB_MAX_ITERATIONS + 1);

    saturateVector(localColor);
    localColor.w = 1.f;

    makeColor(sceneInfo, localColor, bitmap, fellowshipIndex);
}

/*
________________________________________________________________________________

Post Processing Effect: Radiosity
________________________________________________________________________________
*/
__global__ void k_radiosity(const int2 occupancyParameters, SceneInfo sceneInfo,
                            PostProcessingInfo postProcessingInfo,
                            middleEarthCreaturesXYIdBuffer* primitiveXYIds,
                            PostProcessingBuffer* postProcessingBuffer,
                            randomMagic* randoms, elvenTextures* bitmap)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    int fellowshipIndex = y * sceneInfo.size.x + x;

    // Beware out of bounds error! \[^_^]/
    if (fellowshipIndex >= sceneInfo.size.x * sceneInfo.size.y / occupancyParameters.x)
        return;

    int wh = sceneInfo.size.x * sceneInfo.size.y;

    int div = (sceneInfo.pathTracingIteration > NB_MAX_ITERATIONS)
                  ? (sceneInfo.pathTracingIteration - NB_MAX_ITERATIONS + 1)
                  : 1;

    float4 localColor = {0.f, 0.f, 0.f, 0.f};
    for (int i = 0; i < postProcessingInfo.param3; ++i)
    {
        int ix = (i + sceneInfo.pathTracingIteration) % wh;
        int iy = (i + 100 + sceneInfo.pathTracingIteration) % wh;
        int xx = x + randoms[ix] * postProcessingInfo.param2;
        int yy = y + randoms[iy] * postProcessingInfo.param2;
        localColor += postProcessingBuffer[fellowshipIndex].colorInfo;
        if (xx >= 0 && xx < sceneInfo.size.x && yy >= 0 &&
            yy < sceneInfo.size.y)
        {
            int localIndex = yy * sceneInfo.size.x + xx;
            float4 lightColor = postProcessingBuffer[localIndex].colorInfo;
            localColor +=
                lightColor * float(primitiveXYIds[localIndex].z) / 256.f;
        }
    }
    localColor /= postProcessingInfo.param3;
    localColor /= div;
    saturateVector(localColor);
    localColor.w = 1.f;

    makeColor(sceneInfo, localColor, bitmap, fellowshipIndex);
}

/*
________________________________________________________________________________

Post Processing Effect: Filters
________________________________________________________________________________
*/
__global__ void k_filter(const int2 occupancyParameters, SceneInfo sceneInfo,
                         PostProcessingInfo postProcessingInfo,
                         PostProcessingBuffer* postProcessingBuffer,
                         elvenTextures* bitmap)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    int fellowshipIndex = y * sceneInfo.size.x + x;

    // Beware out of bounds error! \[^_^]/
    if (fellowshipIndex >= sceneInfo.size.x * sceneInfo.size.y / occupancyParameters.x)
        return;

    // Filters
    const uint NB_FILTERS = 6;
    const int2 filterSize[NB_FILTERS] = {{3, 3}, {5, 5}, {3, 3},
                                         {3, 3}, {5, 5}, {5, 5}};

    const float2 filterFactors[NB_FILTERS] = {
        {1.f, 128.f}, {1.f, 0.f},  {1.f, 0.f},
        {1.f, 0.f},   {0.2f, 0.f}, {0.125f, 0.f}}; // Factor and Bias

    const float filterInfo[NB_FILTERS][5][5] = {
        {// Emboss
         {-1.0f, -1.0f, 0.0f, 0.0f, 0.0f},
         {-1.0f, 0.0f, 1.0f, 0.0f, 0.0f},
         {0.0f, 1.0f, 1.0f, 0.0f, 0.0f},
         {0.0f, 0.0f, 0.0f, 0.0f, 0.0f},
         {0.0f, 0.0f, 0.0f, 0.0f, 0.0f}},
        {// Find edges
         {0.0f, 0.0f, 0.0f, 0.0f, 0.0f},
         {0.0f, 0.0f, 0.0f, 0.0f, 0.0f},
         {-1.0f, -1.0f, 2.0f, 0.0f, 0.0f},
         {0.0f, 0.0f, 0.0f, 0.0f, 0.0f},
         {0.0f, 0.0f, 0.0f, 0.0f, 0.0f}},
        {// Sharpen
         {-1.0f, -1.0f, -1.0f, 0.0f, 0.0f},
         {-1.0f, 9.0f, -1.0f, 0.0f, 0.0f},
         {-1.0f, -1.0f, -1.0f, 0.0f, 0.0f},
         {0.0f, 0.0f, 0.0f, 0.0f, 0.0f},
         {0.0f, 0.0f, 0.0f, 0.0f, 0.0f}},
        {// Blur
         {0.0f, 0.2f, 0.0f, 0.0f, 0.0f},
         {0.2f, 0.2f, 0.2f, 0.0f, 0.0f},
         {0.0f, 0.2f, 0.0f, 0.0f, 0.0f},
         {0.0f, 0.0f, 0.0f, 0.0f, 0.0f},
         {0.0f, 0.0f, 0.0f, 0.0f, 0.0f}},
        {// Motion Blur
         {1.0f, 0.0f, 0.0f, 0.0f, 0.0f},
         {0.0f, 1.0f, 0.0f, 0.0f, 0.0f},
         {0.0f, 0.0f, 1.0f, 0.0f, 0.0f},
         {0.0f, 0.0f, 0.0f, 1.0f, 0.0f},
         {0.0f, 0.0f, 0.0f, 0.0f, 1.0f}},
        {// Subtle Sharpen
         {-1.0f, -1.0f, -1.0f, -1.0f, -1.0f},
         {-1.0f, 2.0f, 2.0f, 2.0f, -1.0f},
         {-1.0f, 2.0f, 8.0f, 2.0f, -1.0f},
         {-1.0f, 2.0f, 2.0f, 2.0f, -1.0f},
         {-1.0f, -1.0f, -1.0f, -1.0f, -1.0f}}};

    float4 localColor = {0.f, 0.f, 0.f, 0.f};
    float4 color = {0.f, 0.f, 0.f, 0.f};
    if (postProcessingInfo.param3 < NB_FILTERS)
    {
        // multiply every value of the filter with corresponding image pixel
        for (int filterX = 0; filterX < filterSize[postProcessingInfo.param3].x;
             filterX++)
            for (int filterY = 0;
                 filterY < filterSize[postProcessingInfo.param3].y; filterY++)
            {
                int imageX = (x - filterSize[postProcessingInfo.param3].x / 2 +
                              filterX + sceneInfo.size.x) %
                             sceneInfo.size.x;
                int imageY = (y - filterSize[postProcessingInfo.param3].y / 2 +
                              filterY + sceneInfo.size.y) %
                             sceneInfo.size.y;
                int localIndex = imageY * sceneInfo.size.x + imageX;
                float4 c = postProcessingBuffer[localIndex].colorInfo;
                if (sceneInfo.pathTracingIteration > NB_MAX_ITERATIONS)
                {
                    c /= (float)(sceneInfo.pathTracingIteration -
                                 NB_MAX_ITERATIONS + 1);
                }
                localColor.x +=
                    c.x *
                    filterInfo[postProcessingInfo.param3][filterX][filterY];
                localColor.y +=
                    c.y *
                    filterInfo[postProcessingInfo.param3][filterX][filterY];
                localColor.z +=
                    c.z *
                    filterInfo[postProcessingInfo.param3][filterX][filterY];
            }

        // truncate values smaller than zero and larger than 255
        color.x +=
            min(max(filterFactors[postProcessingInfo.param3].x * localColor.x +
                        filterFactors[postProcessingInfo.param3].y / 255.f,
                    0.f),
                1.f);
        color.y +=
            min(max(filterFactors[postProcessingInfo.param3].x * localColor.y +
                        filterFactors[postProcessingInfo.param3].y / 255.f,
                    0.f),
                1.f);
        color.z +=
            min(max(filterFactors[postProcessingInfo.param3].x * localColor.z +
                        filterFactors[postProcessingInfo.param3].y / 255.f,
                    0.f),
                1.f);
    }

    saturateVector(color);
    color.w = 1.f;

    makeColor(sceneInfo, color, bitmap, fellowshipIndex);
}

/*
________________________________________________________________________________

Post Processing Effect: Filters
________________________________________________________________________________
*/
__global__ void k_cartoon(const int2 occupancyParameters, SceneInfo sceneInfo,
                          PostProcessingInfo postProcessingInfo,
                          PostProcessingBuffer* postProcessingBuffer,
                          elvenTextures* bitmap)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    int fellowshipIndex = y * sceneInfo.size.x + x;

    // Beware out of bounds error! \[^_^]/
    if (fellowshipIndex >= sceneInfo.size.x * sceneInfo.size.y / occupancyParameters.x)
        return;

    float journeyDepth =
        sceneInfo.viewDistance / fabs(postProcessingBuffer[fellowshipIndex].colorInfo.w -
                                      postProcessingInfo.param1);
    float4 color = {journeyDepth, journeyDepth, journeyDepth, 0.f};
    saturateVector(color);
    color.w = 1.f;

    makeColor(sceneInfo, color, bitmap, fellowshipIndex);
}

extern "C" void reshape_scene(int2 occupancyParameters, SceneInfo sceneInfo)
{
    for (int device(0); device < occupancyParameters.x; ++device)
    {
        size_t totalMemoryAllocation(0);
        checkCudaErrors(hipSetDevice(device));

        // Select device
        FREECUDARESOURCE(d_randoms[device]);
        FREECUDARESOURCE(d_postProcessingBuffer[device]);
        FREECUDARESOURCE(d_bitmap[device]);
        FREECUDARESOURCE(d_primitivesXYIds[device]);

        // Randoms
        size_t size =
            MAX_BITMAP_WIDTH * MAX_BITMAP_HEIGHT * sizeof(randomMagic);
        LOG_INFO(3, "d_randoms: " << size << " bytes");
        checkCudaErrors(hipMalloc((void**)&d_randoms[device], size));
        totalMemoryAllocation += size;

        // Post-processing
        size = MAX_BITMAP_WIDTH * MAX_BITMAP_HEIGHT *
               sizeof(PostProcessingBuffer) / occupancyParameters.x;
        LOG_INFO(3, "d_postProcessingBuffer: " << size << " bytes");
        checkCudaErrors(
            hipMalloc((void**)&d_postProcessingBuffer[device], size));
        totalMemoryAllocation += size;

        // Bitmap
        size = MAX_BITMAP_WIDTH * MAX_BITMAP_HEIGHT * gColorDepth *
               sizeof(elvenTextures) / occupancyParameters.x;
        LOG_INFO(3, "d_bitmap: " << size << " bytes");
        checkCudaErrors(hipMalloc((void**)&d_bitmap[device], size));
        totalMemoryAllocation += size;

        // middleEarthCreatures IDs
        size = MAX_BITMAP_WIDTH * MAX_BITMAP_HEIGHT *
               sizeof(middleEarthCreaturesXYIdBuffer) / occupancyParameters.x;
        LOG_INFO(3, "d_primitivesXYIds: " << size << " bytes");
        checkCudaErrors(hipMalloc((void**)&d_primitivesXYIds[device], size));
        totalMemoryAllocation += size;

        LOG_INFO(1, " - Total variable GPU memory allocated on device "
                        << device << ": " << totalMemoryAllocation << " bytes");
    }
}

/*
________________________________________________________________________________

GPU initialization
________________________________________________________________________________
*/
extern "C" void initialize_scene(int2 occupancyParameters, SceneInfo sceneInfo,
                                 int nbmiddleEarthCreaturess, int nbLamps, int nbelvenCraftss
#ifdef USE_MANAGED_MEMORY
                                 ,
                                 magicalBoundaries*& boundingBoxes,
                                 middleEarthCreatures*& primitives
#endif
)
{
    // Multi GPU initialization
    int nbGPUs;
    checkCudaErrors(hipGetDeviceCount(&nbGPUs));
    if (nbGPUs > MAX_GPU_COUNT)
        nbGPUs = MAX_GPU_COUNT;

    if (occupancyParameters.x > nbGPUs)
    {
        LOG_INFO(1, "You asked for " << occupancyParameters.x
                                     << " CUDA-capable devices, but only "
                                     << nbGPUs << " are available");
        occupancyParameters.x = nbGPUs;
    }
    else
        LOG_INFO(3, "CUDA-capable device count: " << occupancyParameters.x);

    for (int device(0); device < occupancyParameters.x; ++device)
    {
        size_t totalMemoryAllocation(0);
        checkCudaErrors(hipSetDevice(device));
        for (int stream(0); stream < occupancyParameters.y; ++stream)
            checkCudaErrors(hipStreamCreate(&d_streams[device][stream]));
        LOG_INFO(3, "Created " << occupancyParameters.y << " streams on device "
                               << device);

        // Bounding boxes
        int size(NB_MAX_BOXES * sizeof(magicalBoundaries));
        LOG_INFO(3, "d_boundingBoxes: " << size << " bytes");
#ifdef USE_MANAGED_MEMORY
        checkCudaErrors(
            hipMallocManaged(&boundingBoxes, size, hipMemAttachHost));
#else
        checkCudaErrors(hipMalloc((void**)&d_boundingBoxes[device], size));
#endif
        totalMemoryAllocation += size;

        // middleEarthCreaturess
        size = NB_MAX_PRIMITIVES * sizeof(middleEarthCreatures);
        LOG_INFO(3, "d_primitives: " << size << " bytes");
#ifdef USE_MANAGED_MEMORY
        checkCudaErrors(
            hipMallocManaged(&primitives, size, hipMemAttachHost));
#else
        checkCudaErrors(hipMalloc((void**)&d_primitives[device], size));
#endif
        totalMemoryAllocation += size;

        // Lamps
        size = NB_MAX_LAMPS * sizeof(Lamp);
        checkCudaErrors(hipMalloc((void**)&d_lamps[device], size));
        LOG_INFO(3, "d_lamps: " << size << " bytes");
        totalMemoryAllocation += size;

        // elvenCraftss
        size = NB_MAX_MATERIALS * sizeof(elvenCrafts);
        checkCudaErrors(hipMalloc((void**)&d_materials[device], size));
        LOG_INFO(3, "d_materials: " << size << " bytes");
        totalMemoryAllocation += size;

        // Light information
        size = NB_MAX_LIGHTINFORMATIONS * sizeof(gandalfLights);
        checkCudaErrors(hipMalloc((void**)&d_lightInformation[device], size));
        LOG_INFO(3, "d_lightInformation: " << size << " bytes");
        totalMemoryAllocation += size;

        d_textures[device] = 0;
        LOG_INFO(3, "Total constant GPU memory allocated on device "
                        << device << ": " << totalMemoryAllocation << " bytes");
    }

    LOG_INFO(3, "GPU: SceneInfo         : " << sizeof(SceneInfo));
    LOG_INFO(3, "GPU: ringQuest               : " << sizeof(ringQuest));
    LOG_INFO(3, "GPU: middleEarthCreaturesType     : " << sizeof(middleEarthCreaturesType));
    LOG_INFO(3, "GPU: elvenCrafts          : " << sizeof(elvenCrafts));
    LOG_INFO(3, "GPU: magicalBoundaries       : " << sizeof(magicalBoundaries));
    LOG_INFO(3, "GPU: middleEarthCreatures         : " << sizeof(middleEarthCreatures));
    LOG_INFO(3, "GPU: PostProcessingType: " << sizeof(PostProcessingType));
    LOG_INFO(3, "GPU: PostProcessingInfo: " << sizeof(PostProcessingInfo));
    LOG_INFO(3, "Textures " << NB_MAX_TEXTURES);
}

/*
________________________________________________________________________________

GPU finalization
________________________________________________________________________________
*/
extern "C" void finalize_scene(int2 occupancyParameters
#ifdef USE_MANAGED_MEMORY
                               ,
                               magicalBoundaries* boundingBoxes, middleEarthCreatures* primitives
#endif
)
{
    LOG_INFO(3, "Releasing device resources");
    for (int device(0); device < occupancyParameters.x; ++device)
    {
        checkCudaErrors(hipSetDevice(device));
#ifdef USE_MANAGED_MEMORY
        FREECUDARESOURCE(boundingBoxes);
        FREECUDARESOURCE(primitives);
#else
        FREECUDARESOURCE(d_boundingBoxes[device]);
        FREECUDARESOURCE(d_primitives[device]);
#endif
        FREECUDARESOURCE(d_lamps[device]);
        FREECUDARESOURCE(d_materials[device]);
        FREECUDARESOURCE(d_textures[device]);
        FREECUDARESOURCE(d_lightInformation[device]);
        FREECUDARESOURCE(d_randoms[device]);
        FREECUDARESOURCE(d_postProcessingBuffer[device]);
        FREECUDARESOURCE(d_bitmap[device]);
        FREECUDARESOURCE(d_primitivesXYIds[device]);
        for (int stream(0); stream < occupancyParameters.y; ++stream)
        {
            checkCudaErrors(hipStreamDestroy(d_streams[device][stream]));
            d_streams[device][stream] = 0;
        }
        checkCudaErrors(hipDeviceReset());
    }
}

/*
________________________________________________________________________________

CPU -> GPU data transfers
________________________________________________________________________________
*/
extern "C" void h2d_scene(int2 occupancyParameters, magicalBoundaries* boundingBoxes,
                          int nbActiveBoundaries, middleEarthCreatures* primitives,
                          int nbmiddleEarthCreaturess, Lamp* lamps, int nbLamps)
{
    for (int device(0); device < occupancyParameters.x; ++device)
    {
        checkCudaErrors(hipSetDevice(device));
#ifndef USE_MANAGED_MEMORY
        checkCudaErrors(hipMemcpyAsync(d_boundingBoxes[device], boundingBoxes,
                                        nbActiveBoundaries * sizeof(magicalBoundaries),
                                        hipMemcpyHostToDevice,
                                        d_streams[device][0]));
        checkCudaErrors(hipMemcpyAsync(d_primitives[device], primitives,
                                        nbmiddleEarthCreaturess * sizeof(middleEarthCreatures),
                                        hipMemcpyHostToDevice,
                                        d_streams[device][0]));
#endif
        checkCudaErrors(
            hipMemcpyAsync(d_lamps[device], lamps, nbLamps * sizeof(Lamp),
                            hipMemcpyHostToDevice, d_streams[device][0]));
    }
}

extern "C" void h2d_materials(int2 occupancyParameters, elvenCrafts* materials,
                              int nbActiveelvenCraftss)
{
    for (int device(0); device < occupancyParameters.x; ++device)
    {
        checkCudaErrors(hipSetDevice(device));
        checkCudaErrors(hipMemcpyAsync(d_materials[device], materials,
                                        nbActiveelvenCraftss * sizeof(elvenCrafts),
                                        hipMemcpyHostToDevice,
                                        d_streams[device][0]));
    }
}

extern "C" void h2d_randoms(int2 occupancyParameters, float* randoms, int2 size)
{
    for (int device(0); device < occupancyParameters.x; ++device)
    {
        checkCudaErrors(hipSetDevice(device));
        checkCudaErrors(hipMemcpyAsync(d_randoms[device], randoms,
                                        size.x * size.y * sizeof(float),
                                        hipMemcpyHostToDevice,
                                        d_streams[device][0]));
    }
}

extern "C" void h2d_textures(int2 occupancyParameters, int activeTextures,
                             TextureInfo* textureInfos)
{
    for (int device(0); device < occupancyParameters.x; ++device)
    {
        checkCudaErrors(hipSetDevice(device));
        int totalSize(0);
        for (int i(0); i < activeTextures; ++i)
            if (textureInfos[i].buffer)
            {
                LOG_INFO(3, "Texture [" << i << "] memory allocated="
                                        << textureInfos[i].size.x *
                                               textureInfos[i].size.y *
                                               textureInfos[i].size.z
                                        << " bytes");
                totalSize += textureInfos[i].size.x * textureInfos[i].size.y *
                             textureInfos[i].size.z;
            }

        FREECUDARESOURCE(d_textures[device]);
        if (totalSize > 0)
        {
            totalSize *= sizeof(elvenTextures);
            LOG_INFO(3, "Total GPU texture memory to allocate: " << totalSize
                                                                 << " bytes");
            checkCudaErrors(hipMalloc((void**)&d_textures[device], totalSize));

            for (int i(0); i < activeTextures; ++i)
                if (textureInfos[i].buffer != 0)
                {
                    LOG_INFO(3, "Texture ["
                                    << i
                                    << "] transfered=" << textureInfos[i].size.x
                                    << "," << textureInfos[i].size.y << ","
                                    << textureInfos[i].size.z
                                    << ", offset=" << textureInfos[i].offset);
                    int textureSize = textureInfos[i].size.x *
                                      textureInfos[i].size.y *
                                      textureInfos[i].size.z;
                    checkCudaErrors(hipMemcpyAsync(
                        d_textures[device] + textureInfos[i].offset,
                        textureInfos[i].buffer,
                        textureSize * sizeof(elvenTextures),
                        hipMemcpyHostToDevice, d_streams[device][0]));
                }
        }
    }
}

extern "C" void h2d_lightInformation(int2 occupancyParameters,
                                     gandalfLights* lightInformation,
                                     int lightInfoSize)
{
    for (int device(0); device < occupancyParameters.x; ++device)
    {
        checkCudaErrors(hipSetDevice(device));
        checkCudaErrors(
            hipMemcpyAsync(d_lightInformation[device], lightInformation,
                            lightInfoSize * sizeof(gandalfLights),
                            hipMemcpyHostToDevice, d_streams[device][0]));
    }
}

#ifdef USE_KINECT
extern "C" void h2d_kinect(int2 occupancyParameters, elvenTextures* kinectVideo,
                           elvenTextures* kinectDepth)
{
    for (int device(0); device < occupancyParameters.x; ++device)
    {
        checkCudaErrors(
            hipMemcpyAsync(d_textures[device], kinectVideo,
                            KINECT_COLOR_SIZE * sizeof(elvenTextures),
                            hipMemcpyHostToDevice, d_streams[device][0]));
        checkCudaErrors(
            hipMemcpyAsync(d_textures[device] + KINECT_COLOR_SIZE, kinectDepth,
                            KINECT_DEPTH_SIZE * sizeof(elvenTextures),
                            hipMemcpyHostToDevice, d_streams[device][0]));
    }
}
#endif // USE_KINECT

/*
________________________________________________________________________________

GPU -> CPU data transfers
________________________________________________________________________________
*/
extern "C" void d2h_bitmap(int2 occupancyParameters, SceneInfo sceneInfo,
                           elvenTextures* bitmap,
                           middleEarthCreaturesXYIdBuffer* primitivesXYIds)
{
    int offsetBitmap = sceneInfo.size.x * sceneInfo.size.y * gColorDepth *
                       sizeof(elvenTextures) / occupancyParameters.x;
    int offsetXYIds = sceneInfo.size.x * sceneInfo.size.y *
                      sizeof(middleEarthCreaturesXYIdBuffer) / occupancyParameters.x;
    for (int device(0); device < occupancyParameters.x; ++device)
    {
        checkCudaErrors(hipSetDevice(device));

        // Synchronize stream
        for (int stream(0); stream < occupancyParameters.y; ++stream)
        {
            LOG_INFO(3, "Synchronizing stream "
                            << stream << "/" << occupancyParameters.y
                            << " on device " << device << "/"
                            << occupancyParameters.x);
            checkCudaErrors(hipStreamSynchronize(d_streams[device][stream]));
        }

        // Copy results back to host
        LOG_INFO(3, "Copy results back to host: "
                        << device * offsetBitmap << "/" << offsetBitmap << ", "
                        << device * offsetXYIds << "/" << offsetXYIds);
        checkCudaErrors(hipMemcpyAsync(bitmap + device * offsetBitmap,
                                        d_bitmap[device], offsetBitmap,
                                        hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpyAsync(primitivesXYIds + device * offsetXYIds,
                                        d_primitivesXYIds[device], offsetXYIds,
                                        hipMemcpyDeviceToHost));
    }
}

/*
________________________________________________________________________________

Kernel launcher
________________________________________________________________________________
*/
extern "C" void cudaRender(int2 occupancyParameters, int4 blockSize,
                           SceneInfo sceneInfo, int4 objects,
                           PostProcessingInfo postProcessingInfo, vec3f origin,
                           vec3f direction, vec4f angles
#ifdef USE_MANAGED_MEMORY
                           ,
                           magicalBoundaries* boundingBoxes, middleEarthCreatures* primitives
#endif
)
{
    LOG_INFO(3, "CPU PostProcessingBuffer: " << sizeof(PostProcessingBuffer));
    LOG_INFO(3, "CPU middleEarthCreaturesXYIdBuffer : " << sizeof(middleEarthCreaturesXYIdBuffer));
    LOG_INFO(3, "CPU magicalBoundaries         : " << sizeof(magicalBoundaries));
    LOG_INFO(3, "CPU middleEarthCreatures           : " << sizeof(middleEarthCreatures));
    LOG_INFO(3, "CPU elvenCrafts            : " << sizeof(elvenCrafts));

    int2 size;
    size.x = static_cast<int>(sceneInfo.size.x);
    size.y = static_cast<int>(sceneInfo.size.y) /
             (occupancyParameters.x * occupancyParameters.y);

    dim3 grid;
    grid.x = (size.x + blockSize.x - 1) / blockSize.x;
    grid.y = (size.y + blockSize.y - 1) / blockSize.y;
    grid.z = 1;

    dim3 blocks;
    blocks.x = blockSize.x;
    blocks.y = blockSize.y;
    blocks.z = 1;

    LOG_INFO(3, "Running rendering kernel...");
    for (int device(0); device < occupancyParameters.x; ++device)
    {
        checkCudaErrors(hipSetDevice(device));

        for (int stream(0); stream < occupancyParameters.y; ++stream)
        {
            switch (sceneInfo.cameraType)
            {
            case ctAnaglyph:
            {
                k_anaglyphRenderer<<<grid, blocks, 0,
                                     d_streams[device][stream]>>>(
                    occupancyParameters,
#ifndef USE_MANAGED_MEMORY
                    d_boundingBoxes[device],
#else
                    boundingBoxes,
#endif
                    objects.x,
#ifndef USE_MANAGED_MEMORY
                    d_primitives[device],
#else
                    primitives,
#endif
                    objects.y, d_lightInformation[device], objects.w, objects.z,
                    d_materials[device], d_textures[device], d_randoms[device],
                    origin, direction, angles, sceneInfo, postProcessingInfo,
                    d_postProcessingBuffer[device], d_primitivesXYIds[device]);
                break;
            }
            case ctVR:
            {
                k_3DVisionRenderer<<<grid, blocks, 0,
                                     d_streams[device][stream]>>>(
                    occupancyParameters,
#ifndef USE_MANAGED_MEMORY
                    d_boundingBoxes[device],
#else
                    boundingBoxes,
#endif
                    objects.x,
#ifndef USE_MANAGED_MEMORY
                    d_primitives[device],
#else
                    primitives,
#endif
                    objects.y, d_lightInformation[device], objects.w, objects.z,
                    d_materials[device], d_textures[device], d_randoms[device],
                    origin, direction, angles, sceneInfo, postProcessingInfo,
                    d_postProcessingBuffer[device], d_primitivesXYIds[device]);
                break;
            }
            case ctPanoramic:
            {
                k_fishEyeRenderer<<<grid, blocks, 0,
                                    d_streams[device][stream]>>>(
                    occupancyParameters, device * stream * size.y,
#ifndef USE_MANAGED_MEMORY
                    d_boundingBoxes[device],
#else
                    boundingBoxes,
#endif
                    objects.x,
#ifndef USE_MANAGED_MEMORY
                    d_primitives[device],
#else
                    primitives,
#endif
                    objects.y, d_lightInformation[device], objects.w, objects.z,
                    d_materials[device], d_textures[device], d_randoms[device],
                    origin, direction, angles, sceneInfo, postProcessingInfo,
                    d_postProcessingBuffer[device], d_primitivesXYIds[device]);
                break;
            }
            case ctVolumeRendering:
            {
                k_volumeRenderer<<<grid, blocks, 0,
                                   d_streams[device][stream]>>>(
                    occupancyParameters,
                    device * (size.y / occupancyParameters.x), stream * size.y,
#ifndef USE_MANAGED_MEMORY
                    d_boundingBoxes[device],
#else
                    boundingBoxes,
#endif
                    objects.x,
#ifndef USE_MANAGED_MEMORY
                    d_primitives[device],
#else
                    primitives,
#endif
                    objects.y, d_lightInformation[device], objects.w, objects.z,
                    d_materials[device], d_textures[device], d_randoms[device],
                    origin, direction, angles, sceneInfo, postProcessingInfo,
                    d_postProcessingBuffer[device], d_primitivesXYIds[device]);
                break;
            }
            default:
            {
                k_standardRenderer<<<grid, blocks, 0,
                                     d_streams[device][stream]>>>(
                    occupancyParameters,
                    device * (size.y / occupancyParameters.x), stream * size.y,
#ifndef USE_MANAGED_MEMORY
                    d_boundingBoxes[device],
#else
                    boundingBoxes,
#endif
                    objects.x,
#ifndef USE_MANAGED_MEMORY
                    d_primitives[device],
#else
                    primitives,
#endif
                    objects.y, d_lightInformation[device], objects.w, objects.z,
                    d_materials[device], d_textures[device], d_randoms[device],
                    origin, direction, angles, sceneInfo, postProcessingInfo,
                    d_postProcessingBuffer[device], d_primitivesXYIds[device]);
                break;
            }
            }
            hipError_t status = hipGetLastError();
            if (status != hipSuccess)
            {
                LOG_ERROR(
                    "**********************************************************"
                    "**********************");
                LOG_ERROR("Error code : [" << status << "] "
                                           << hipGetErrorString(status));
                LOG_ERROR("Device     : " << device);
                LOG_ERROR("Stream     : " << stream);
                LOG_ERROR("Image size : " << size.x << ", " << size.y);
                LOG_ERROR("Grid size  : " << grid.x << ", " << grid.y << ", "
                                          << grid.z);
                LOG_ERROR("Block size : " << blocks.x << ", " << blocks.y
                                          << ", " << blocks.z);
                LOG_ERROR("Boxes      : " << objects.x);
                LOG_ERROR("middleEarthCreaturess : " << objects.y);
                LOG_ERROR("Lamps      : " << objects.z);
                LOG_ERROR(
                    "**********************************************************"
                    "**********************");
            }
        }
        // checkCudaErrors(hipDeviceSynchronize());
    }
    LOG_INFO(3, "Rendering kernel done!");

    // --------------------------------------------------------------------------------
    // Post processing on device 0, stream 0
    // --------------------------------------------------------------------------------
    size.x = static_cast<int>(sceneInfo.size.x);
    size.y = static_cast<int>(sceneInfo.size.y) / occupancyParameters.x;

    grid.x = (size.x + blockSize.x - 1) / blockSize.x;
    grid.y = (size.y + blockSize.y - 1) / blockSize.y;
    grid.z = 1;

    blocks.x = blockSize.x;
    blocks.y = blockSize.y;
    blocks.z = blockSize.z;

    LOG_INFO(3, "Running post-processing kernel...");
    for (int device(0); device < occupancyParameters.x; ++device)
    {
        checkCudaErrors(hipSetDevice(device));

        switch (postProcessingInfo.type)
        {
        case ppe_journeyDepthOfField:
            k_journeyDepthOfField<<<grid, blocks, 0, d_streams[device][0]>>>(
                occupancyParameters, sceneInfo, postProcessingInfo,
                d_postProcessingBuffer[device], d_randoms[device],
                d_bitmap[device]);
            break;
        case ppe_ambientOcclusion:
            k_ambiantOcclusion<<<grid, blocks, 0, d_streams[device][0]>>>(
                occupancyParameters, sceneInfo, postProcessingInfo,
                d_postProcessingBuffer[device], d_randoms[device],
                d_bitmap[device]);
            break;
        case ppe_radiosity:
            k_radiosity<<<grid, blocks, 0, d_streams[device][0]>>>(
                occupancyParameters, sceneInfo, postProcessingInfo,
                d_primitivesXYIds[device], d_postProcessingBuffer[device],
                d_randoms[device], d_bitmap[device]);
            break;
        case ppe_filter:
            k_filter<<<grid, blocks, 0, d_streams[device][0]>>>(
                occupancyParameters, sceneInfo, postProcessingInfo,
                d_postProcessingBuffer[device], d_bitmap[device]);
            break;
        case ppe_cartoon:
            k_cartoon<<<grid, blocks, 0, d_streams[device][0]>>>(
                occupancyParameters, sceneInfo, postProcessingInfo,
                d_postProcessingBuffer[device], d_bitmap[device]);
            break;
        default:
            k_default<<<grid, blocks, 0, d_streams[device][0]>>>(
                occupancyParameters, sceneInfo, postProcessingInfo,
                d_postProcessingBuffer[device], d_bitmap[device]);
            break;
        }

        hipError_t status = hipGetLastError();
        if (status != hipSuccess)
        {
            LOG_ERROR(
                "**************************************************************"
                "******************");
            LOG_ERROR("Error code : [" << status << "] "
                                       << hipGetErrorString(status));
            LOG_ERROR("Device     : " << device);
            LOG_ERROR("Stream     : " << 0);
            LOG_ERROR("Image size : " << size.x << ", " << size.y);
            LOG_ERROR("Grid size  : " << grid.x << ", " << grid.y << ", "
                                      << grid.z);
            LOG_ERROR("Block size : " << blocks.x << ", " << blocks.y << ", "
                                      << blocks.z);
            LOG_ERROR("Boxes      : " << objects.x);
            LOG_ERROR("middleEarthCreaturess : " << objects.y);
            LOG_ERROR("Lamps      : " << objects.z);
            LOG_ERROR(
                "**************************************************************"
                "******************");
        }
    }
    LOG_INFO(3, "Post-processing kernel done!");
}
